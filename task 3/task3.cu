#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <vector>

using namespace std;


__global__ void distanceKernel(int* a, int m, int n, int* d)
{
    int row1 = blockIdx.x * blockDim.x + threadIdx.x;
    int row2 = blockIdx.y * blockDim.y + threadIdx.y;
    if (row1 < m && row2 < m)
    {
        int tmp = 0;
        for (int i = 0; i < n; i++)
        {
            tmp += (a[row1 * n + i] - a[row2 * n + i]) * (a[row1 * n + i] - a[row2 * n + i]);
        }
        d[row1 * m + row2] = tmp;
    }
}

void distance(int* a, int m, int n, int* d)
{
    dim3 threadsPerBlock(m, m);
    dim3 blocksPerGrid(1, 1);
    if (m > 16) {
        threadsPerBlock.x = 16;
        threadsPerBlock.y = 16;
        blocksPerGrid.x = ceil(double(m) / double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(m) / double(threadsPerBlock.y));
        printf("%d\n", blocksPerGrid.x);
    }

    distanceKernel << <blocksPerGrid, threadsPerBlock >> > (a, m, n, d);
    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
            printf("kernel launch failed with error \"%s\".\n",
                hipGetErrorString(cudaerr));
    }
}

int main(int argc, char* argv[])
{
    int devID;
    hipDeviceProp_t deviceProps;

    printf("[%s] - Starting...\n", argv[0]);

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char**)argv);

    // get device name
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s]\n", deviceProps.name);

    //int m = 1024;
    //int n = 512;
    int m = 128;
    int n = 16;
    int nmbytes = n * m * sizeof(int);
    int mmbytes = m * m * sizeof(int);

    // allocate host memory
    int* a = 0;
    checkCudaErrors(hipHostMalloc((void**)&a, nmbytes));
    memset(a, 0, nmbytes);
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
            a[i * m + j] = 6;
    }

    int* d = 0;
    checkCudaErrors(hipHostMalloc((void**)&d, mmbytes));
    memset(d, 0, mmbytes);


    // allocate device memory
    int* d_a = 0;
    checkCudaErrors(hipMalloc((void**)&d_a, nmbytes));
    checkCudaErrors(hipMemset(d_a, 255, nmbytes));

    int* d_d = 0;
    checkCudaErrors(hipMalloc((void**)&d_d, mmbytes));
    checkCudaErrors(hipMemset(d_d, 255, mmbytes));

    // set kernel launch configuration
    //dim3 threads = dim3(512, 1);
    //dim3 blocks  = dim3(m * n / threads.x, 1);

    // create cuda event handles
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    StopWatchInterface* timer = NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);

    checkCudaErrors(hipDeviceSynchronize());
    float gpu_time = 0.0f;

    // asynchronously issue work to the GPU (all to stream 0)
    sdkStartTimer(&timer);
    hipEventRecord(start, 0);

    hipMemcpyAsync(d_a, a, nmbytes, hipMemcpyHostToDevice, 0);
    hipMemcpyAsync(d_d, d, mmbytes, hipMemcpyHostToDevice, 0);

    distance(d_a, m, n, d_d);
    hipMemcpyAsync(a, d_a, nmbytes, hipMemcpyDeviceToHost, 0);
    hipMemcpyAsync(d, d_d, mmbytes, hipMemcpyDeviceToHost, 0);
    hipEventRecord(stop, 0);
    sdkStopTimer(&timer);

    // have CPU do some work while waiting for stage 1 to finish
    unsigned long long counter = 0;

    while (hipEventQuery(stop) == hipErrorNotReady)
    {
        counter++;
    }

    checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));

    // print the cpu and gpu times
    printf("time spent executing by the GPU: %.2f\n", gpu_time);
    printf("time spent by CPU in CUDA calls: %.2f\n", sdkGetTimerValue(&timer));
    printf("CPU executed %lu iterations while waiting for GPU to finish\n", counter);

    // check the output for correctness
    //bool bFinalResults = correct_output(a, n, value);

    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < m; j++)
            printf("%d ", d[i * m + j]);
        printf("\n");
    }

    // release resources
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipHostFree(a));
    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipHostFree(d));
    checkCudaErrors(hipFree(d_d));

    return 0;
}